#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------------
*
*    This file is part of the UltraCold project.
*
*    UltraCold is free software: you can redistribute it and/or modify
*    it under the terms of the GNU General Public License as published by
*    the Free Software Foundation, either version 3 of the License, or
*    any later version.
*    UltraCold is distributed in the hope that it will be useful,
*    but WITHOUT ANY WARRANTY; without even the implied warranty of
*    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*    GNU General Public License for more details.
*    You should have received a copy of the GNU General Public License
*    along with UltraCold.  If not, see <https://www.gnu.org/licenses/>.
*
*--------------------------------------------------------------------------------*/

#include "cudaGPSolver.cuh"
#include "mesh_fourier_space.hpp"
#include "DataOut.hpp"
#include "hipcub/hipcub.hpp"
#include "hipfft/hipfft.h"
#include "kernel_utilities.cuh"

#define PI 3.1415926535897932384626433
#define TWOPI (2*PI)

namespace UltraCold
{
    namespace cudaSolvers
    {

        /**
         * @brief Constructor for 1d problems
         * */

        GPSolver::GPSolver(Vector<double> &x,
                           Vector<std::complex<double>> &psi_0,
                           Vector<double> &Vext,
                           double scattering_length)
        {

            // Check that the order of the Vectors provided is correctly 1
            if(psi_0.order() != 1 || Vext.order() != 1)
            {
                std::cout
                        << "\n\n"
                        << "**************************************************************************\n"
                        << "Error found in the constructor of a (cuda) GPSolver for a Gross-Pitaevskii\n"
                        << "equation in one space dimension. The orders of the Vectors provided as\n"
                        << "input are not consistent. In particular, initial wave function and external\n"
                        << "potential provided are not 1-dimensional.\n"
                        << "Terminating the program now...\n"
                        << "**************************************************************************\n"
                        << "\n\n"
                        <<
                        std::endl;
                exit(1);
            }
            problem_is_1d=true;

            // Check the dimensions of the Vectors provided are consistent
            nx = x.extent(0);
            ny = 0;
            nz = 0;
            npoints=nx;
            if(psi_0.extent(0) != nx || Vext.extent(0) != nx)
            {
                std::cout
                        << "\n\n"
                        << "**************************************************************************\n"
                        << "Error found in the constructor of a (cuda) GPSolver for a Gross-Pitaevskii\n"
                        << "equation in one space dimension. The dimensions of the Vectors provided as\n"
                        << "input are not consistent.\n"
                        << "Terminating the program now...\n"
                        << "**************************************************************************\n"
                        << "\n\n"
                        <<
                        std::endl;
                exit(1);
            }

            // Initialize the thread grid, i.e. choose the number of cuda threads per block and the number of blocks.
            blockSize = 512;
            gridSize = (npoints + blockSize - 1) / blockSize;

            // Allocate memory for all device arrays
            hipMalloc(&external_potential_d,nx*sizeof(double));
            hipMalloc(&kmod2_d,             nx*sizeof(double));
            hipMalloc(&density_d,           nx*sizeof(double));
            hipMalloc(&wave_function_d,     nx*sizeof(hipDoubleComplex));
            hipMalloc(&hpsi_d,              nx*sizeof(hipDoubleComplex));
            hipMalloc(&ft_wave_function_d,  nx*sizeof(hipDoubleComplex));

            // Allocate space for device and managed scalars
            hipMalloc(&scattering_length_d,sizeof(double));
            hipMallocManaged(&norm_d,              sizeof(double));
            hipMallocManaged(&initial_norm_d,      sizeof(double));
            hipMallocManaged(&chemical_potential_d,sizeof(double));

            // Get the first necessary copies of input data from host to device
            hipMemcpy(external_potential_d,Vext.data(),       npoints*sizeof(double),         hipMemcpyHostToDevice);
            hipMemcpy(wave_function_d,     psi_0.data(),      npoints*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
            hipMemcpy(scattering_length_d, &scattering_length,1      *sizeof(double),         hipMemcpyHostToDevice);

            // Initialize the mesh in Fourier space, and copy it to the device
            Vector<double> kx(nx);
            Vector<double> kmod2(nx);
            create_mesh_in_Fourier_space(x,kx);
            for (size_t i = 0; i < nx; ++i) kmod2(i) = std::pow(kx(i),2);
            hipMemcpy(kmod2_d,  kmod2.data(),npoints*sizeof(double),hipMemcpyHostToDevice);

            // Initialize space steps
            dx = x(1)-x(0);
            dv = dx;

            // Initialize the device reduce kernel
            hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,density_d,norm_d,npoints);
            hipDeviceSynchronize();

            // Allocate temporary storage memory, required for reduction kernels
            hipMalloc(&temporary_storage_d,size_temporary_storage);
            hipDeviceSynchronize();

            // Calculate initial norm
            calculate_density(density_d,wave_function_d,npoints);
            hipDeviceSynchronize();
            hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,density_d,norm_d,npoints);
            hipDeviceSynchronize();
            norm_d[0]=norm_d[0]*dv;
            initial_norm_d[0]=norm_d[0];
            std::cout << "Initial norm: " << initial_norm_d[0] << std::endl;

            // Initialize the wave function to return as a result
            result_wave_function.reinit(nx);

            // Initialize the host and device vectors containing the mesh axis. This can be useful in particular for
            // data output
            x_axis.reinit(nx);
            x_axis=x;
            kx_axis.reinit(nx);
            kx_axis=kx;
            hipMalloc(&x_axis_d,nx*sizeof(double));
            hipMemcpy(x_axis_d,x_axis.data(),nx*sizeof(double),hipMemcpyHostToDevice);
            hipMalloc(&kx_axis_d,nx*sizeof(double));
            hipMemcpy(kx_axis_d,kx_axis.data(),nx*sizeof(double),hipMemcpyHostToDevice);
            r2mod.reinit(nx);
            for(int i = 0; i < nx; ++i)
                r2mod[i] = std::pow(x(i),2);
            hipMalloc(&r2mod_d,nx*sizeof(double));
            hipMemcpy(r2mod_d,r2mod.data(),nx*sizeof(double),hipMemcpyHostToDevice);

        }

        /**
         * @brief Constructor for 2d problems
         */

        GPSolver::GPSolver(Vector<double> &x,
                           Vector<double> &y,
                           Vector<std::complex<double>> &psi_0,
                           Vector<double> &Vext,
                           double scattering_length)
        {

            // Check that the order of the Vectors provided is correctly 1
            if(psi_0.order() != 2 || Vext.order() != 2)
            {
                std::cout
                        << "\n\n"
                        << "**************************************************************************\n"
                        << "Error found in the constructor of a (cuda) GPSolver for a Gross-Pitaevskii\n"
                        << "equation in two space dimension. The orders of the Vectors provided as\n"
                        << "input are not consistent. In particular, initial wave function and external\n"
                        << "potential provided are not 2-dimensional.\n"
                        << "Terminating the program now...\n"
                        << "**************************************************************************\n"
                        << "\n\n"
                        <<
                        std::endl;
                exit(1);
            }
            problem_is_2d=true;

            // Check the dimensions of the Vectors provided are consistent
            nx = x.extent(0);
            ny = y.extent(0);
            nz = 0;
            npoints=nx*ny;
            if(psi_0.extent(0) != nx || Vext.extent(0) != nx ||
               psi_0.extent(1) != ny || Vext.extent(1) != ny)
            {
                std::cout
                        << "\n\n"
                        << "**************************************************************************\n"
                        << "Error found in the constructor of a (cuda) GPSolver for a Gross-Pitaevskii\n"
                        << "equation in two space dimension. The dimensions of the Vectors provided as\n"
                        << "input are not consistent.\n"
                        << "Terminating the program now...\n"
                        << "**************************************************************************\n"
                        << "\n\n"
                        <<
                        std::endl;
                exit(1);
            }

            // Initialize the thread grid, i.e. choose the number of cuda threads per block and the number of blocks.
            blockSize = 512;
            gridSize = (npoints + blockSize - 1) / blockSize;

            // Allocate memory for all device arrays
            hipMalloc(&external_potential_d,npoints*sizeof(double));
            hipMalloc(&kmod2_d,             npoints*sizeof(double));
            hipMalloc(&density_d,           npoints*sizeof(double));
            hipMalloc(&wave_function_d,     npoints*sizeof(hipDoubleComplex));
            hipMalloc(&hpsi_d,              npoints*sizeof(hipDoubleComplex));
            hipMalloc(&ft_wave_function_d,  npoints*sizeof(hipDoubleComplex));

            // Allocate space for device and managed scalars
            hipMalloc(&scattering_length_d,sizeof(double));
            hipMallocManaged(&norm_d,              sizeof(double));
            hipMallocManaged(&initial_norm_d,      sizeof(double));
            hipMallocManaged(&chemical_potential_d,sizeof(double));

            // Get the first necessary copies of input data from host to device
            hipMemcpy(external_potential_d,Vext.data(),       npoints*sizeof(double),         hipMemcpyHostToDevice);
            hipMemcpy(wave_function_d,     psi_0.data(),      npoints*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
            hipMemcpy(scattering_length_d, &scattering_length,1      *sizeof(double),         hipMemcpyHostToDevice);

            // Initialize the mesh in Fourier space, and copy it to the device
            Vector<double> kx(nx);
            Vector<double> ky(ny);
            Vector<double> kmod2(nx,ny);
            create_mesh_in_Fourier_space(x,y,kx,ky);
            for (size_t i = 0; i < nx; ++i)
                for (size_t j = 0; j < ny; ++j)
                    kmod2(i,j) = std::pow(kx(i),2) +
                                 std::pow(ky(j),2);
            hipMemcpy(kmod2_d,kmod2.data(),npoints*sizeof(double),hipMemcpyHostToDevice);

            // Initialize space steps
            dx = x(1)-x(0);
            dy = y(1)-y(0);
            dv = dx*dy;

            // Initialize the device reduce kernel
            hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,density_d,norm_d,npoints);
            hipDeviceSynchronize();

            // Allocate temporary storage memory, required for reduction kernels
            hipMalloc(&temporary_storage_d,size_temporary_storage);
            hipDeviceSynchronize();

            // Calculate initial norm
            calculate_density(density_d,wave_function_d,npoints);
            hipDeviceSynchronize();
            hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,density_d,norm_d,npoints);
            hipDeviceSynchronize();
            norm_d[0]=norm_d[0]*dv;
            initial_norm_d[0]=norm_d[0];
            std::cout << "Initial norm: " << initial_norm_d[0] << std::endl;

            // Initialize the wave function to return as a result
            result_wave_function.reinit(nx,ny);

            // Initialize the host vectors containing the mesh axis. This can be useful in particular for data output
            x_axis.reinit(nx);
            y_axis.reinit(ny);
            x_axis=x;
            y_axis=y;
            kx_axis.reinit(nx);
            ky_axis.reinit(ny);
            kx_axis=kx;
            ky_axis=ky;
            hipMalloc(&x_axis_d,nx*sizeof(double));
            hipMalloc(&y_axis_d,ny*sizeof(double));
            hipMemcpy(x_axis_d,x_axis.data(),nx*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(y_axis_d,y_axis.data(),ny*sizeof(double),hipMemcpyHostToDevice);
            hipMalloc(&kx_axis_d,nx*sizeof(double));
            hipMalloc(&ky_axis_d,ny*sizeof(double));
            hipMemcpy(kx_axis_d,kx_axis.data(),nx*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(ky_axis_d,ky_axis.data(),ny*sizeof(double),hipMemcpyHostToDevice);
            r2mod.reinit(nx,ny);
            for(int i = 0; i < nx; ++i)
                for(int j = 0; j < ny; ++j)
                    r2mod(i,j) = std::pow(x(i),2)+std::pow(y(j),2);
            hipMalloc(&r2mod_d,npoints*sizeof(double));
            hipMemcpy(r2mod_d,r2mod.data(),npoints*sizeof(double),hipMemcpyHostToDevice);
        }

        /**
         * @brief Constructor for 3d problems
         */

        GPSolver::GPSolver(Vector<double> &x,
                           Vector<double> &y,
                           Vector<double> &z,
                           Vector<std::complex<double>> &psi_0,
                           Vector<double> &Vext,
                           double scattering_length)
        {

            // Check that the order of the Vectors provided is correctly 1
            if(psi_0.order() != 3 || Vext.order() != 3)
            {
                std::cout
                        << "\n\n"
                        << "**************************************************************************\n"
                        << "Error found in the constructor of a (cuda) GPSolver for a Gross-Pitaevskii\n"
                        << "equation in three space dimension. The orders of the Vectors provided as\n"
                        << "input are not consistent. In particular, initial wave function and external\n"
                        << "potential provided are not 3-dimensional.\n"
                        << "Terminating the program now...\n"
                        << "**************************************************************************\n"
                        << "\n\n"
                        <<
                        std::endl;
                exit(1);
            }
            problem_is_3d=true;

            // Check the dimensions of the Vectors provided are consistent
            nx = x.extent(0);
            ny = y.extent(0);
            nz = z.extent(0);
            npoints=nx*ny*nz;
            if(psi_0.extent(0) != nx || Vext.extent(0) != nx ||
               psi_0.extent(1) != ny || Vext.extent(1) != ny ||
               psi_0.extent(2) != nz || Vext.extent(2) != nz)
            {
                std::cout
                        << "\n\n"
                        << "**************************************************************************\n"
                        << "Error found in the constructor of a (cuda) GPSolver for a Gross-Pitaevskii\n"
                        << "equation in three space dimension. The dimensions of the Vectors provided as\n"
                        << "input are not consistent.\n"
                        << "Terminating the program now...\n"
                        << "**************************************************************************\n"
                        << "\n\n"
                        <<
                        std::endl;
                exit(1);
            }

            // Initialize the thread grid, i.e. choose the number of cuda threads per block and the number of blocks.
            blockSize = 512;
            gridSize = (npoints + blockSize - 1) / blockSize;

            // Allocate memory for all device arrays
            hipMalloc(&external_potential_d,npoints*sizeof(double));
            hipMalloc(&kmod2_d,             npoints*sizeof(double));
            hipMalloc(&density_d,           npoints*sizeof(double));
            hipMalloc(&wave_function_d,     npoints*sizeof(hipDoubleComplex));
            hipMalloc(&hpsi_d,              npoints*sizeof(hipDoubleComplex));
            hipMalloc(&ft_wave_function_d,  npoints*sizeof(hipDoubleComplex));

            // Allocate space for device and managed scalars
            hipMalloc(&scattering_length_d,sizeof(double));
            hipMallocManaged(&norm_d,              sizeof(double));
            hipMallocManaged(&initial_norm_d,      sizeof(double));
            hipMallocManaged(&chemical_potential_d,sizeof(double));

            // Get the first necessary copies of input data from host to device
            hipMemcpy(external_potential_d,Vext.data(),       npoints*sizeof(double),         hipMemcpyHostToDevice);
            hipMemcpy(wave_function_d,     psi_0.data(),      npoints*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
            hipMemcpy(scattering_length_d, &scattering_length,1      *sizeof(double),         hipMemcpyHostToDevice);

            // Initialize the mesh in Fourier space, and copy it to the device
            Vector<double> kx(nx);
            Vector<double> ky(ny);
            Vector<double> kz(nz);
            Vector<double> kmod2(nx,ny,nz);
            create_mesh_in_Fourier_space(x,y,z,kx,ky,kz);
            for (size_t i = 0; i < nx; ++i)
                for (size_t j = 0; j < ny; ++j)
                    for (size_t k = 0; k < nz; ++k)
                        kmod2(i,j,k) = std::pow(kx(i),2)+
                                       std::pow(ky(j),2)+
                                       std::pow(kz(k),2);
            hipMemcpy(kmod2_d, kmod2.data(),npoints*sizeof(double),hipMemcpyHostToDevice);

            // Initialize space steps
            dx = x(1)-x(0);
            dy = y(1)-y(0);
            dz = z(1)-z(0);
            dv = dx*dy*dz;

            // Initialize the device reduce kernel
            hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,density_d,norm_d,npoints);
            hipDeviceSynchronize();

            // Allocate temporary storage memory, required for reduction kernels
            hipMalloc(&temporary_storage_d,size_temporary_storage);
            hipDeviceSynchronize();

            // Calculate initial norm
            calculate_density(density_d,wave_function_d,npoints);
            hipDeviceSynchronize();
            hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,density_d,norm_d,npoints);
            hipDeviceSynchronize();
            norm_d[0]=norm_d[0]*dv;
            initial_norm_d[0]=norm_d[0];
            std::cout << "Initial norm: " << initial_norm_d[0] << std::endl;

            // Initialize the wave function to return as a result
            result_wave_function.reinit(nx,ny,nz);

            // Initialize the host vectors containing the mesh axis. This can be useful in particular for data output
            x_axis.reinit(nx);
            y_axis.reinit(ny);
            z_axis.reinit(nz);
            x_axis=x;
            y_axis=y;
            z_axis=z;
            kx_axis.reinit(nx);
            ky_axis.reinit(ny);
            kz_axis.reinit(nz);
            kx_axis=kx;
            ky_axis=ky;
            kz_axis=kz;
            hipMalloc(&x_axis_d,nx*sizeof(double));
            hipMalloc(&y_axis_d,ny*sizeof(double));
            hipMalloc(&z_axis_d,nz*sizeof(double));
            hipMemcpy(x_axis_d,x_axis.data(),nx*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(y_axis_d,y_axis.data(),ny*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(z_axis_d,z_axis.data(),nz*sizeof(double),hipMemcpyHostToDevice);
            hipMalloc(&kx_axis_d,nx*sizeof(double));
            hipMalloc(&ky_axis_d,ny*sizeof(double));
            hipMalloc(&kz_axis_d,nz*sizeof(double));
            hipMemcpy(kx_axis_d,kx_axis.data(),nx*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(ky_axis_d,ky_axis.data(),ny*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(kz_axis_d,kz_axis.data(),nz*sizeof(double),hipMemcpyHostToDevice);
            r2mod.reinit(nx,ny,nz);
            for(int i = 0; i < nx; ++i)
                for(int j = 0; j < ny; ++j)
                    for(int k = 0; k < nz; ++k)
                        r2mod(i,j,k) = std::pow(x(i),2)+std::pow(y(j),2)+std::pow(z(k),2);
            hipMalloc(&r2mod_d,npoints*sizeof(double));
            hipMemcpy(r2mod_d,r2mod.data(),npoints*sizeof(double),hipMemcpyHostToDevice);

	    // Initialize a vector to output the wave function
	    wave_function_output.reinit(nx,ny,nz);
        }

        /**
         * @brief Destructor frees device memory
         *
         */

        GPSolver::~GPSolver()
        {
            hipFree(external_potential_d);
            hipFree(density_d);
            hipFree(norm_d);
            hipFree(initial_norm_d);
            hipFree(wave_function_d);
            hipFree(ft_wave_function_d);
            hipFree(hpsi_d);
            hipFree(x_axis_d);
            hipFree(y_axis_d);
            hipFree(z_axis_d);
            hipFree(kx_axis_d);
            hipFree(ky_axis_d);
            hipFree(kz_axis_d);
            hipFree(kmod2_d);
            hipFree(r2mod_d);
            hipFree(chemical_potential_d);
            hipFree(scattering_length_d);
            hipFree(alpha_d);
            hipFree(beta_d);
            hipFree(time_step_d);
            hipFree(temporary_storage_d);

        }


        /**
         * @brief Calculate the density profile
         *
         * */

        void GPSolver::calculate_density(double *density, hipDoubleComplex *wave_function,int size)
        {
            KernelUtilities::square_vector<<<gridSize,blockSize>>>(density,wave_function,size);
        }

        /**
         *
         * @brief Run the gradient descent
         *
         * \warning No check of the residual!
         *
         */

        std::tuple<Vector<std::complex<double>>, double>
                GPSolver::run_gradient_descent(int max_num_iter,
                                               double alpha,
                                               double beta,
                                               std::ostream &output_stream,
                                               int write_output_every)

        {
            // Initialize the fft plan required for the calculation of the laplacian
            hipfftHandle ft_plan;
            if(problem_is_1d)
                hipfftPlan1d(&ft_plan,nx,HIPFFT_Z2Z,1);
            else if(problem_is_2d)
                hipfftPlan2d(&ft_plan,nx,ny,HIPFFT_Z2Z);
            else if(problem_is_3d)
                hipfftPlan3d(&ft_plan,nx,ny,nz,HIPFFT_Z2Z);

            //--------------------------------------------------//
            //    Here the gradient-descent iterations start    //
            //--------------------------------------------------//

            // Allocate space for some new data on the device
            hipMalloc(&alpha_d,sizeof(double));
            hipMemcpy(alpha_d,&alpha,sizeof(double),hipMemcpyHostToDevice);
            hipMalloc(&beta_d,sizeof(double));
            hipMemcpy(beta_d,&beta,sizeof(double),hipMemcpyHostToDevice);
            hipDoubleComplex* psi_new;
            hipDoubleComplex* psi_old;
            hipMalloc(&psi_new,npoints*sizeof(hipDoubleComplex));
            hipMalloc(&psi_old,npoints*sizeof(hipDoubleComplex));

            // Loop starts here
            for (int it = 0; it < max_num_iter; ++it)
            {

                // Calculate the action of the laplacian
                hipfftExecZ2Z(ft_plan, wave_function_d, ft_wave_function_d, HIPFFT_FORWARD);
                hipDeviceSynchronize();
                KernelUtilities::vector_multiplication<<<gridSize,blockSize>>>(ft_wave_function_d,kmod2_d,npoints);
                hipDeviceSynchronize();
                hipfftExecZ2Z(ft_plan, ft_wave_function_d, hpsi_d, HIPFFT_BACKWARD);
                hipDeviceSynchronize();
                KernelUtilities::rescale<<<gridSize,blockSize>>>(hpsi_d,0.5*pow(TWOPI,2)/npoints,npoints);
                hipDeviceSynchronize();

                // Calculate the rest of H|psi>
                KernelUtilities::step_2_hpsi<<<gridSize,blockSize>>>(hpsi_d,
                                                                    wave_function_d,
                                                                    external_potential_d,
                                                                    scattering_length_d,
                                                                    npoints);
                hipDeviceSynchronize();

                // Perform a gradient descent (plus heavy-ball) step
                KernelUtilities::gradient_descent_step<<<gridSize,blockSize>>>(wave_function_d,
                                                                               hpsi_d,
                                                                               psi_new,
                                                                               psi_old,
                                                                               alpha_d,
                                                                               beta_d,
                                                                               npoints);
                hipDeviceSynchronize();

                // Normalize the wave function
                KernelUtilities::square_vector<<<gridSize,blockSize>>>(density_d,psi_new,npoints);
                hipDeviceSynchronize();
                hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,density_d,norm_d,npoints);
                hipDeviceSynchronize();
                norm_d[0] = norm_d[0]*dv;
                KernelUtilities::rescale<<<gridSize,blockSize>>>(wave_function_d,
                                                                 psi_new,
                                                                 sqrt(initial_norm_d[0]/norm_d[0]),
                                                                 npoints);
                hipDeviceSynchronize();

                // Calculate the chemical potential
                KernelUtilities::vector_multiplication<<<gridSize,blockSize>>>(density_d,hpsi_d,wave_function_d,npoints);
                hipDeviceSynchronize();
                hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,density_d,chemical_potential_d,npoints);
                hipDeviceSynchronize();
                chemical_potential_d[0] = chemical_potential_d[0]*dv/norm_d[0];

                // Eventually write some output
                if(it % write_output_every == 0)
                    write_gradient_descent_output(it);

            }

            // Free the remaining arrays from the device
            hipFree(psi_new);
            hipFree(psi_old);

            // Copy out the results
            hipMemcpy(result_wave_function.data(),
                       wave_function_d,
                       npoints*sizeof(hipDoubleComplex),
                       hipMemcpyDeviceToHost);
            double result_chemical_potential = chemical_potential_d[0];

            // Return
            return std::make_pair(result_wave_function,result_chemical_potential);

        }

        /**
         *
         * @brief Write gradient descent output
         *
         * */

        void GPSolver::write_gradient_descent_output(int it)
        {
            std::cout << it << " " << chemical_potential_d[0] << std::endl;
        }

        /**
         * @brief Real-time operator splitting
         * */

        void GPSolver::run_operator_splitting(int number_of_time_steps, double time_step, std::ostream &output_stream,
                                              int write_output_every)
        {
            // Copy input data into the device
            hipMallocManaged(&time_step_d,sizeof(double));
            hipMemcpy(time_step_d,&time_step,sizeof(double),hipMemcpyHostToDevice);

            // Initialize the fft plan required for the calculation of the laplacian
            hipfftHandle ft_plan;
            if(problem_is_1d)
                hipfftPlan1d(&ft_plan,nx,HIPFFT_Z2Z,1);
            else if(problem_is_2d)
                hipfftPlan2d(&ft_plan,nx,ny,HIPFFT_Z2Z);
            else if(problem_is_3d)
                hipfftPlan3d(&ft_plan,nx,ny,nz,HIPFFT_Z2Z);

            //----------------------------------------------------//
            //    Here the operator-splitting iterations start    //
            //----------------------------------------------------//

            for (size_t it = 0; it < number_of_time_steps; ++it)
            {

                // Write output starting from the very first iteration
                if(it % write_output_every == 0)
                {
                    hipMemcpy(wave_function_output.data(),
                               wave_function_d,
                               npoints*sizeof(hipDoubleComplex),
                               hipMemcpyDeviceToHost);
                    write_operator_splitting_output(it,output_stream);
                }

                // Solve step-1 of operator splitting, i.e. the one NOT involving Fourier transforms
                KernelUtilities::step_1_operator_splitting<<<gridSize,blockSize>>>(wave_function_d,
                                                                                   external_potential_d,
                                                                                   time_step_d,
                                                                                   scattering_length_d,
                                                                                   npoints);
                hipDeviceSynchronize();

                // Solve step-2 of operator splitting, i.e. the one actually involving Fourier transforms
                hipfftExecZ2Z(ft_plan,wave_function_d,ft_wave_function_d,HIPFFT_FORWARD);
                hipDeviceSynchronize();
                KernelUtilities::aux_step_2_operator_splitting<<<gridSize,blockSize>>>(ft_wave_function_d,
                                                                                       kmod2_d,
                                                                                       time_step_d,
                                                                                       npoints);
                hipDeviceSynchronize();
                hipfftExecZ2Z(ft_plan,ft_wave_function_d,wave_function_d,HIPFFT_BACKWARD);
                hipDeviceSynchronize();
                KernelUtilities::rescale<<<gridSize,blockSize>>>(wave_function_d,1./npoints,npoints);
                hipDeviceSynchronize();
            hipMemcpy(wave_function_output.data(),
                       wave_function_d,
                       npoints*sizeof(hipDoubleComplex),
                       hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            RealSpaceOutput::DataOut psi_out;
            psi_out.set_output_name("psi_step2"+std::to_string(it/write_output_every));
            psi_out.write_slice1d_csv(x_axis,wave_function_output,"x");

            }

        }

        /**
         *
         * @brief Operator splitting output.
         *
         * This function is called after a copy of the current wave function outside of the GPU, is such a way that it
         * can be used for example for data analysis or to write it to a file for visualization. Since each call
         * blocks the real-time evolution on the GPU until the function has finished, it is better to use it with
         * moderation to avoid a big loss of performance.
         *
         */

        void GPSolver::write_operator_splitting_output(int it,std::ostream& output_stream)
        {}

        /**
         *
         * @brief Reinitialize the solver with a new external potential and wave function.
         *
         * */

        void GPSolver::reinit(Vector<std::complex<double>> &psi, Vector<double> &Vext)
        {

            hipMemcpy(wave_function_d,psi.data(),npoints*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
            hipMemcpy(external_potential_d,Vext.data(),npoints*sizeof(double),hipMemcpyHostToDevice);

        }
    }
}

#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------------
*
*    This file is part of the UltraCold project.
*
*    UltraCold is free software: you can redistribute it and/or modify
*    it under the terms of the GNU General Public License as published by
*    the Free Software Foundation, either version 3 of the License, or
*    any later version.
*    UltraCold is distributed in the hope that it will be useful,
*    but WITHOUT ANY WARRANTY; without even the implied warranty of
*    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*    GNU General Public License for more details.
*    You should have received a copy of the GNU General Public License
*    along with UltraCold.  If not, see <https://www.gnu.org/licenses/>.
*
*--------------------------------------------------------------------------------*/

#include "kernel_utilities.cuh"
#define PI 3.1415926535897932384626433
#define TWOPI (2*PI)

namespace UltraCold
{
    namespace KernelUtilities
    {

        ///////////////////////////////////////////////////////
        // Pure device kernels
        ///////////////////////////////////////////////////////

        // A useful complex exponential function
        __device__ hipDoubleComplex complex_exponential(hipDoubleComplex input)
        {
            hipDoubleComplex res;
            double t = expf (input.x);
            sincos (input.y, &res.y, &res.x);
            res.x *= t;
            res.y *= t;
            return res;
        }

        ////////////////////////////////////////////////////
        // Global kernels
        ////////////////////////////////////////////////////

        /**
         *
         * @brief Calculate the square of a complex vector, storing the result in another complex vector
         *
         * */

        __global__ void square_vector(hipDoubleComplex* result,
                                      hipDoubleComplex* input,
                                      int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i].x = input[i].x*input[i].x +
                              input[i].y*input[i].y;
                result[i].y = 0.0;
            }
        }
        /**
         *
         * @brief Calculate the square of a complex vector
         *
         * */

        __global__ void square_vector(double* result,
                                      hipDoubleComplex* input,
                                      int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i] = input[i].x*input[i].x +
                            input[i].y*input[i].y;
            }
        }

        /**
         *
         * @brief Calculate the square of a real vector
         *
         * */

        __global__ void square_vector(double* result,
                                      double* input,
                                      int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i] = input[i]*input[i];
            }
        }

        /**
         *
         * @brief Multiply a real vector times the square of a complex vector
         *
         * */

        __global__ void vector_average(double* result,
                                       double* input1,
                                       hipDoubleComplex* input2,
                                       int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i] = input1[i]*(input2[i].x*input2[i].x+input2[i].y*input2[i].y);
            }
        }

        /**
         *
         * @brief Multiply two complex vectors. Overwrite the first one
         *
         * */

        __global__ void vector_multiplication(hipDoubleComplex* result,
                                              hipDoubleComplex* input,
                                              int size)
        {
            hipDoubleComplex temp;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                temp = result[i];
                result[i].x = temp.x*input[i].x - temp.y*input[i].y;
                result[i].y = temp.y*input[i].x + temp.x*input[i].y;
            }
        }

        /**
         *
         * @brief Multiply a complex and a real vector. Overwrite the complex one
         *
         * */

        __global__ void vector_multiplication(hipDoubleComplex* result,
                                              double* input,
                                              int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i].x = result[i].x*input[i];
                result[i].y = result[i].y*input[i];
            }
        }

        /**
          *
          * @brief Multiply two real vectors. Overwrite the first one
          *
          * */

        __global__ void vector_multiplication(double* result,
                                              double* input,
                                              int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i] = result[i]*input[i];
            }
        }

        /**
         *
         * @brief Multiply two complex vectors in the case in which the result is a real one
         *
         * */

        __global__ void vector_multiplication(double* result,
                                              hipDoubleComplex* input1,
                                              hipDoubleComplex* input2,
                                              int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i] = input1[i].x * input2[i].x;
            }
        }

        /**
         *
         * @brief Second step in the operator splitting method for gradient descent
         *
         * */

        __global__ void step_2_hpsi(hipDoubleComplex* hpsi,
                                    hipDoubleComplex* psi,
                                    double* Vext,
                                    double* scattering_length,
                                    int size)
        {

            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                hpsi[i].x = hpsi[i].x +
                            (Vext[i]
                            + 4*PI*scattering_length[0]*(psi[i].x*psi[i].x+psi[i].y*psi[i].y)
                            ) *
                            psi[i].x;
                hpsi[i].y = hpsi[i].y +
                            (Vext[i]
                             + 4*PI*scattering_length[0]*(psi[i].x*psi[i].x+psi[i].y*psi[i].y)
                            ) *
                            psi[i].y;
            }
        }

        /**
         *
         * @brief Second step in the operator splitting method for gradient descent for dipolars
         *
         * */

        __global__ void step_2_dipolar_hpsi(hipDoubleComplex* hpsi,
                                            hipDoubleComplex* psi,
                                            double* Vext,
                                            hipDoubleComplex* Phi_dd,
                                            double* scattering_length,
                                            double* gamma_epsilon_dd,
                                            int size)
        {

            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            double aux=0.0;
            for (int i = index; i < size; i += stride)
            {
                aux= hipCabs(psi[i]);
                hpsi[i].x = hpsi[i].x +
                            (Vext[i]
                            + 4*PI*scattering_length[0]*pow(aux,2)
                            + Phi_dd[i].x
                            + gamma_epsilon_dd[0]*pow(aux,3)
                            ) *
                            psi[i].x;
                hpsi[i].y = hpsi[i].y +
                            (Vext[i]
                             + 4*PI*scattering_length[0]*pow(aux,2)
                             + Phi_dd[i].x
                             + gamma_epsilon_dd[0]*pow(aux,3)
                            ) *
                            psi[i].y;
            }
        }

        /**
         *
         * @brief Rescale a vector for a given input scalar
         *
         * */

        __global__ void rescale(hipDoubleComplex* result,
                                double input,
                                int size)
        {

            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i].x = result[i].x * input;
                result[i].y = result[i].y * input;
            }

        }

        /**
         *
         * @brief Rescale a vector for a given input scalar, storing the result in another vector
         *
         * */

        __global__ void rescale(hipDoubleComplex* result,
                                hipDoubleComplex* input1,
                                double input2,
                                int size)
        {

            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i].x = input1[i].x * input2;
                result[i].y = input1[i].y * input2;
            }

        }

        /**
         * @brief Gradient descent plus heavy-ball step
         *
         * */

        __global__ void gradient_descent_step(hipDoubleComplex* psi,
                                              hipDoubleComplex* hpsi,
                                              hipDoubleComplex* psi_new,
                                              hipDoubleComplex* psi_old,
                                              double* alpha,
                                              double* beta,
                                              int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                psi_new[i].x = (1.0 + beta[0])*psi[i].x - alpha[0]*hpsi[i].x - beta[0]*psi_old[i].x;
                psi_old[i].x = psi[i].x;
                psi_new[i].y = (1.0 + beta[0])*psi[i].y - alpha[0]*hpsi[i].y - beta[0]*psi_old[i].y;
                psi_old[i].y = psi[i].y;
            }
        }

        /**
         * @brief Solve step-1 operator splitting
         *
         * */

        __global__ void step_1_operator_splitting(hipDoubleComplex* psi,
                                                  double* Vext,
                                                  double* time_step,
                                                  double* scattering_length,
                                                  int size)
        {
            hipDoubleComplex aux;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                aux.x = 0.0;
                aux.y = - time_step[0] * (Vext[i] + 4*PI*scattering_length[0]*(psi[i].x*psi[i].x+psi[i].y*psi[i].y) );
                psi[i] = hipCmul(psi[i],complex_exponential(aux));
            }
        }

        /**
         * @brief Solve step-1 operator splitting for dipolars
         *
         * */

        __global__ void step_1_operator_splitting_dipolars(hipDoubleComplex* psi,
                                                           double* Vext,
                                                           hipDoubleComplex* Phi_dd,
                                                           double* time_step,
                                                           double* scattering_length,
                                                           double* gamma_epsilon_dd,
                                                           int size)
        {
            hipDoubleComplex aux;
            double aux2;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                aux2 = hipCabs(psi[i]);
                aux.x = 0.0;
                aux.y = - time_step[0] * (Vext[i]
                                          + 4*PI*scattering_length[0]*pow(aux2,2)
                                          + Phi_dd[i].x
                                          + gamma_epsilon_dd[0]*pow(aux2,3)
                                          );
                psi[i] = hipCmul(psi[i],complex_exponential(aux));
            }
        }

        /**
         *
         * @brief A useful help for step-2 of operator splitting
         *
         * */

        __global__ void aux_step_2_operator_splitting(hipDoubleComplex* psitilde,
                                                      double* kmod2,
                                                      double* time_step,
                                                      int size)
        {
            hipDoubleComplex aux;
	    aux.x = 0.0;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                aux.y = - 0.5 * time_step[0] * pow(TWOPI,2) * kmod2[i];
                psitilde[i] = hipCmul(psitilde[i],complex_exponential(aux));
            }
        }
    }
}
